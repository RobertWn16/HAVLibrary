#include "hip/hip_runtime.h"
#include "ColorSpaceConversion.cuh"
#include "LUTS.cuh"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <cuda/std/cmath>
#include <hip/hip_fp16.h>
#include <>

constexpr float SDR_NITS = 80.0f;

//ACES curve constants
__device__ float aces_a = 2.51f;
__device__ float aces_b = 0.03f;
__device__ float aces_c = 2.43f;
__device__ float aces_d = 0.59f;
__device__ float aces_e = 0.14f;

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
	return x < lower ? lower : (x > upper ? upper : x);
}

__device__ float __device_RGB_XYZ_Mat[3][3];
__device__ float __device_XYZ_RGB_Mat[3][3];

__device__ inline unsigned short float2half(float f)
{
	union fasi { float f; unsigned int i; };
	fasi v;
	v.f = f;

	unsigned short sign = (v.i >> 31) & 0x1;
	short exp = ((v.i >> 23) & 0xff) - 127;
	unsigned short mant = (v.i >> 13) & 0x3ff;

	if (exp < -14)
	{
		exp = 0;
		mant = (0x400 & mant) >> (-exp - 14);
	}
	else if (exp < 16)
	{
		exp += 15;
	}
	else
	{
		// just make it inf, ignore NaN
		exp = 31;
		mant = 0;
	}

	return (sign << 15) | (exp << 10) | mant;
}

__global__ void p016_HDR10_bgra64_HDR10_PQ_ACES_kernel(unsigned short* cuLuma,
	unsigned short* cuChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float max_content_luminance,
	float display_luminance,
	float XYZ_RGB_Matrix[3][3],
	float RGB_XYZ_Matrix[3][3],
	float wr,
	float wb,
	float wg,
	float wgb,
	float wgr,
	float wr_coef,
	float wb_coef,
	float white_black_coeff,
	unsigned short* destImage,
	bool exAlpha,
	unsigned int alpha)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
	int resolution = width * heigth;
	unsigned int pitchFactor = 3;


	float display_lum_coeff = display_luminance / SDR_NITS;
	unsigned int shift_depth = 6;
	unsigned int Y_offset = 64;
	unsigned int UV_offset = 512;
	int maxValue = 1023.0f;

	int Y = 0.0f;
	int U = 0.0f;
	int V = 0.0f;

	float X_plane = 0.0f;
	float Y_plane = 0.0f;
	float Z_plane = 0.0f;

	float R_FP32 = 0.0f;
	float G_FP32 = 0.0f;
	float B_FP32 = 0.0f;
	half A_FP32 = 0.0f;
	if (exAlpha) {
		pitchFactor = 4;
		A_FP32 = (float)alpha;
	}

	unsigned int curRow = 0;
	unsigned int curColumn = 0;
	unsigned int u_idx = 0;
	unsigned int v_idx = 0;

	for (int i = index; i < resolution; i += stride)
	{
		curRow = i / width;
		curColumn = i - curRow * width;
		u_idx = curRow / 2 * width + curColumn;
		u_idx -= u_idx & 1;
		v_idx = u_idx + 1;

		Y = (cuLuma[i] >> shift_depth) - Y_offset;
		U = (cuChroma[u_idx] >> shift_depth) - UV_offset;
		V = (cuChroma[v_idx] >> shift_depth) - UV_offset;

		R_FP32 = (Y + wr_coef * V) * white_black_coeff;
		B_FP32 = (Y + wb_coef * U) * white_black_coeff;
		G_FP32 = ((Y - wr * R_FP32 - wb * B_FP32) / wg) * white_black_coeff;

		R_FP32 = R_FP32 < 0 ? 0 : (R_FP32 > (maxValue - 1) ? (maxValue - 1) : R_FP32);
		G_FP32 = G_FP32 < 0 ? 0 : (G_FP32 > (maxValue - 1) ? (maxValue - 1) : G_FP32);
		B_FP32 = B_FP32 < 0 ? 0 : (B_FP32 > (maxValue - 1) ? (maxValue - 1) : B_FP32);

		R_FP32 = EOTF_LUT[(int)R_FP32];
		G_FP32 = EOTF_LUT[(int)G_FP32];
		B_FP32 = EOTF_LUT[(int)B_FP32];

		R_FP32 = R_FP32 < 0 ? 0 : (R_FP32 > (maxValue - 1) ? (maxValue - 1) : R_FP32);
		G_FP32 = G_FP32 < 0 ? 0 : (G_FP32 > (maxValue - 1) ? (maxValue - 1) : G_FP32);
		B_FP32 = B_FP32 < 0 ? 0 : (B_FP32 > (maxValue - 1) ? (maxValue - 1) : B_FP32);

		X_plane = __device_RGB_XYZ_Mat[0][0] * R_FP32 + __device_RGB_XYZ_Mat[0][1] * G_FP32 + __device_RGB_XYZ_Mat[0][2] * B_FP32;
		Y_plane = __device_RGB_XYZ_Mat[1][0] * R_FP32 + __device_RGB_XYZ_Mat[1][1] * G_FP32 + __device_RGB_XYZ_Mat[1][2] * B_FP32;
		Z_plane = __device_RGB_XYZ_Mat[2][0] * R_FP32 + __device_RGB_XYZ_Mat[2][1] * G_FP32 + __device_RGB_XYZ_Mat[2][2] * B_FP32;

		R_FP32 = __device_XYZ_RGB_Mat[0][0] * X_plane + __device_XYZ_RGB_Mat[0][1] * Y_plane + __device_XYZ_RGB_Mat[0][2] * Z_plane;
		G_FP32 = __device_XYZ_RGB_Mat[1][0] * X_plane + __device_XYZ_RGB_Mat[1][1] * Y_plane + __device_XYZ_RGB_Mat[1][2] * Z_plane;
		B_FP32 = __device_XYZ_RGB_Mat[2][0] * X_plane + __device_XYZ_RGB_Mat[2][1] * Y_plane + __device_XYZ_RGB_Mat[2][2] * Z_plane;

		R_FP32 = (R_FP32 * max_content_luminance) / SDR_NITS;
		G_FP32 = (G_FP32 * max_content_luminance) / SDR_NITS;
		B_FP32 = (B_FP32 * max_content_luminance) / SDR_NITS;

		//3ms
		R_FP32 = R_FP32 * (aces_a * R_FP32 + aces_b) / (R_FP32 * (aces_c * R_FP32 + aces_d) + aces_e) * display_lum_coeff;
		G_FP32 = G_FP32 * (aces_a * G_FP32 + aces_b) / (G_FP32 * (aces_c * G_FP32 + aces_d) + aces_e) * display_lum_coeff;
		B_FP32 = B_FP32 * (aces_a * B_FP32 + aces_b) / (B_FP32 * (aces_c * B_FP32 + aces_d) + aces_e) * display_lum_coeff;

		destImage[pitchFactor * i + 0] = __float2half(R_FP32).operator __half_raw().x;
		destImage[pitchFactor * i + 1] = __float2half(G_FP32).operator __half_raw().x;
		destImage[pitchFactor * i + 2] = __float2half(B_FP32).operator __half_raw().x;

		if (exAlpha)
			destImage[pitchFactor * i + 3] = A_FP32.operator __half_raw().x;
	}
	return;
}

__device__ float ApplySRGBCurve_Fast(float x)
{
	return x < 0.0031308 ? 12.92 * x : 1.13005 * sqrt(x - 0.00228) - 0.13448 * x + 0.005719;
}
__global__ void p016_HDR10_bgra64_HDR10_PQ_Reinhard_kernel(unsigned short* cuLuma,
	unsigned short* cuChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float max_content_luminance,
	float display_luminance,
	float wr,
	float wb,
	float wg,
	float wgb,
	float wgr,
	float wr_coef,
	float wb_coef,
	float white_black_coeff,
	unsigned short* destImage,
	bool exAlpha,
	unsigned int alpha
)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
	int resolution = width * heigth;
	unsigned int pitchFactor = 3;


	float display_lum_coeff = display_luminance / SDR_NITS;
	unsigned int shift_depth = 6;
	unsigned int Y_offset = 64;
	unsigned int UV_offset = 512;
	int maxValue = 1023.0f;

	int Y = 0.0f;
	int U = 0.0f;
	int V = 0.0f;

	float X_plane = 0.0f;
	float Y_plane = 0.0f;
	float Z_plane = 0.0f;

	float R_FP32 = 0.0f;
	float G_FP32 = 0.0f;
	float B_FP32 = 0.0f;
	half A_FP32 = 0.0f;
	if (exAlpha) {
		pitchFactor = 4;
		A_FP32 = (float)alpha;
	}

	unsigned int curRow = 0;
	unsigned int curColumn = 0;
	unsigned int u_idx = 0;
	unsigned int v_idx = 0;

	for (int i = index; i < resolution; i += stride)
	{
		curRow = i / width;
		curColumn = i - curRow * width;
		u_idx = curRow / 2 * width + curColumn;
		u_idx -= u_idx & 1;
		v_idx = u_idx + 1;

		Y = (cuLuma[i] >> shift_depth) - Y_offset;
		U = (cuChroma[u_idx] >> shift_depth) - UV_offset;
		V = (cuChroma[v_idx] >> shift_depth) - UV_offset;

		R_FP32 = (Y + wr_coef * V) * white_black_coeff;
		B_FP32 = (Y + wb_coef * U) * white_black_coeff;
		G_FP32 = ((Y - wr * R_FP32 - wb * B_FP32) / wg) * white_black_coeff;

		R_FP32 = R_FP32 < 0 ? 0 : (R_FP32 > (maxValue - 1) ? (maxValue - 1) : R_FP32);
		G_FP32 = G_FP32 < 0 ? 0 : (G_FP32 > (maxValue - 1) ? (maxValue - 1) : G_FP32);
		B_FP32 = B_FP32 < 0 ? 0 : (B_FP32 > (maxValue - 1) ? (maxValue - 1) : B_FP32);

		R_FP32 = EOTF_LUT[(int)R_FP32];
		G_FP32 = EOTF_LUT[(int)G_FP32];
		B_FP32 = EOTF_LUT[(int)B_FP32];

		X_plane = __device_RGB_XYZ_Mat[0][0] * R_FP32 + __device_RGB_XYZ_Mat[0][1] * G_FP32 + __device_RGB_XYZ_Mat[0][2] * B_FP32;
		Y_plane = __device_RGB_XYZ_Mat[1][0] * R_FP32 + __device_RGB_XYZ_Mat[1][1] * G_FP32 + __device_RGB_XYZ_Mat[1][2] * B_FP32;
		Z_plane = __device_RGB_XYZ_Mat[2][0] * R_FP32 + __device_RGB_XYZ_Mat[2][1] * G_FP32 + __device_RGB_XYZ_Mat[2][2] * B_FP32;

		R_FP32 = __device_XYZ_RGB_Mat[0][0] * X_plane + __device_XYZ_RGB_Mat[0][1] * Y_plane + __device_XYZ_RGB_Mat[0][2] * Z_plane;
		G_FP32 = __device_XYZ_RGB_Mat[1][0] * X_plane + __device_XYZ_RGB_Mat[1][1] * Y_plane + __device_XYZ_RGB_Mat[1][2] * Z_plane;
		B_FP32 = __device_XYZ_RGB_Mat[2][0] * X_plane + __device_XYZ_RGB_Mat[2][1] * Y_plane + __device_XYZ_RGB_Mat[2][2] * Z_plane;

		R_FP32 = (R_FP32 * max_content_luminance) / SDR_NITS;
		G_FP32 = (G_FP32 * max_content_luminance) / SDR_NITS;
		B_FP32 = (B_FP32 * max_content_luminance) / SDR_NITS;

		R_FP32 = (R_FP32 / (R_FP32 + 1)) * display_lum_coeff;
		G_FP32 = (G_FP32 / (G_FP32 + 1)) * display_lum_coeff;
		B_FP32 = (B_FP32 / (B_FP32 + 1)) * display_lum_coeff;

		destImage[pitchFactor * i + 0] = __float2half(R_FP32).operator __half_raw().x;
		destImage[pitchFactor * i + 1] = __float2half(G_FP32).operator __half_raw().x;
		destImage[pitchFactor * i + 2] = __float2half(B_FP32).operator __half_raw().x;

		if (exAlpha)
			destImage[pitchFactor * i + 3] = A_FP32.operator __half_raw().x;
	}
	return;
}

__global__ void p016_HDR10_bgra64_HDR10_Linear_kernel(unsigned short* cuLuma,
	unsigned short* cuChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float wr,
	float wb,
	float wg,
	float wgb,
	float wgr,
	float wr_coef,
	float wb_coef,
	float display_luminance,
	float white_black_coeff,
	unsigned short* destImage,
	bool exAlpha,
	unsigned int alpha
)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
	int resolution = width * heigth;
	unsigned int pitchFactor = 3;

	unsigned int shift_depth = 6;
	unsigned int Y_offset = 64;
	unsigned int UV_offset = 512;
	float display_lum_coeff = display_luminance / SDR_NITS;
	int maxValue = 1023.0f;

	int Y = 0.0f;
	int U = 0.0f;
	int V = 0.0f;

	float R_FP32 = 0.0f;
	float G_FP32 = 0.0f;
	float B_FP32 = 0.0f;
	half A_FP32 = 0.0f;
	if (exAlpha) {
		pitchFactor = 4;
		A_FP32 = (float)alpha;
	}

	unsigned int curRow = 0;
	unsigned int curColumn = 0;
	unsigned int u_idx = 0;
	unsigned int v_idx = 0;

	for (int i = index; i < resolution; i += stride)
	{
		curRow = i / width;
		curColumn = i - curRow * width;
		u_idx = curRow / 2 * width + curColumn;
		u_idx -= u_idx & 1;
		v_idx = u_idx + 1;

		Y = (cuLuma[i] >> shift_depth) - Y_offset;
		U = (cuChroma[u_idx] >> shift_depth) - UV_offset;
		V = (cuChroma[v_idx] >> shift_depth) - UV_offset;

		R_FP32 = (Y + wr_coef * V) * white_black_coeff;
		B_FP32 = (Y + wb_coef * U) * white_black_coeff;
		G_FP32 = ((Y - wr * R_FP32 - wb * B_FP32) / wg) * white_black_coeff;

		R_FP32 = (Clamp(R_FP32, 0.0f, (float)maxValue) / maxValue) * display_lum_coeff;
		G_FP32 = (Clamp(G_FP32, 0.0f, (float)maxValue) / maxValue) * display_lum_coeff;
		B_FP32 = (Clamp(B_FP32, 0.0f, (float)maxValue) / maxValue) * display_lum_coeff;

		destImage[pitchFactor * i + 0] = float2half(R_FP32);
		destImage[pitchFactor * i + 1] = float2half(G_FP32);
		destImage[pitchFactor * i + 2] = float2half(B_FP32);

		if (exAlpha)
			destImage[pitchFactor * i + 3] = A_FP32.operator __half_raw().x;
	}

	return;
}

__global__ void p016_HDR10_bgra32_SDR_Linear_kernel(unsigned short* cuLuma,
	unsigned short* cuChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float wr,
	float wb,
	float wg,
	float wgb,
	float wgr,
	float wr_coef,
	float wb_coef,
	float white_black_coeff,
	unsigned char* destImage,
	bool exAlpha,
	unsigned int alpha
)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
	int resolution = width * heigth;
	unsigned int pitch = width;
	unsigned int pitchFactor = 3;

	unsigned int shift_depth = 8;
	unsigned int Y_offset = 16;
	unsigned int UV_offset = 128;
	if (exAlpha)
		pitchFactor = 4;
	
	int maxValue = 255;

	int Y = 0;
	int U = 0;
	int V = 0;

	float R_FP32 = 0.0f;
	float G_FP32 = 0.0f;
	float B_FP32 = 0.0f;

	for (int i = index; i < resolution; i += stride)
	{
		unsigned int curRow = i / width;
		unsigned int curColumn = i - curRow * width;
		unsigned int u_idx = curRow / 2 * width + curColumn;
		u_idx -= u_idx & 1;
		unsigned int v_idx = u_idx + 1;


		Y = (cuLuma[i] >> shift_depth) - Y_offset;
		U = (cuChroma[u_idx] >> shift_depth) - UV_offset;
		V = (cuChroma[v_idx] >> shift_depth) - UV_offset;

		R_FP32 = (Y + wr_coef * V) * white_black_coeff;
		B_FP32 = (Y + wb_coef * U) * white_black_coeff;
		G_FP32 = ((Y - wr * R_FP32 - wb * B_FP32) / wg) * white_black_coeff;

		if (inverted) {
			destImage[pitchFactor * i + 0] = Clamp((int)R_FP32, 0, maxValue); // Limited RGB Saturation
			destImage[pitchFactor * i + 1] = Clamp((int)G_FP32, 0, maxValue);
			destImage[pitchFactor * i + 2] = Clamp((int)B_FP32, 0, maxValue);
		}
		else {
			destImage[pitchFactor * i + 2] = Clamp((int)R_FP32, 0, maxValue); // Limited RGB Saturation
			destImage[pitchFactor * i + 1] = Clamp((int)G_FP32, 0, maxValue);
			destImage[pitchFactor * i + 0] = Clamp((int)B_FP32, 0, maxValue);
		}

		if (exAlpha)
			destImage[pitchFactor * i + 3] = alpha;
	}

	return;
}

__global__ void nv12_SDR_bgra32_SDR_kernel(unsigned char* cuLuma,
	unsigned char* cuChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float wr,
	float wb,
	float wg,
	float wgb,
	float wgr,
	float wr_coef,
	float wb_coef,
	float white_black_coeff,
	unsigned char* destImage,
	bool exAlpha,
	unsigned int alpha
)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
	int resolution = width * heigth;
	unsigned int pitch = width;
	unsigned int pitchFactor = 3;

	if (exAlpha)
		pitchFactor = 4;
	int maxValue = 255.0f;

	int Y = 0;
	int U = 0;
	int V = 0;

	float R_FP32 = 0.0f;
	float G_FP32 = 0.0f;
	float B_FP32 = 0.0f;

	for (int i = index; i < resolution; i += stride)
	{
		unsigned int curRow = i / width;
		unsigned int curColumn = i - curRow * width;
		unsigned int u_idx = curRow / 2 * width + curColumn;
		u_idx -= u_idx & 1;
		unsigned int v_idx = u_idx + 1;


		Y = (int)cuLuma[i] - 16;
		U = (int)cuChroma[u_idx] - 128;
		V = (int)cuChroma[v_idx] - 128;

		R_FP32 = (Y + wr_coef * V) * white_black_coeff;
		B_FP32 = (Y + wb_coef * U) * white_black_coeff;
		G_FP32 = ((Y - wr * R_FP32 - wb * B_FP32) / wg) * white_black_coeff;
	
		if (inverted) {
			destImage[pitchFactor * i + 0] = Clamp((int)R_FP32, 0, maxValue); // Limited RGB Saturation
			destImage[pitchFactor * i + 1] = Clamp((int)G_FP32, 0, maxValue);
			destImage[pitchFactor * i + 2] = Clamp((int)B_FP32, 0, maxValue);
		}
		else {
			destImage[pitchFactor * i + 2] = Clamp((int)R_FP32, 0, maxValue); // Limited RGB Saturation
			destImage[pitchFactor * i + 1] = Clamp((int)G_FP32, 0, maxValue);
			destImage[pitchFactor * i + 0] = Clamp((int)B_FP32, 0, maxValue);
		}

		if (exAlpha)
			destImage[pitchFactor * i + 3] = alpha;
	}

	return;
}

__global__ void bgr24_bgra32_kernel(unsigned char* srcImage,
	unsigned int width,
	unsigned int heigth,
	unsigned int alpha,
	unsigned char* destImage
)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
	unsigned int resolution = width * heigth;
	for (unsigned int i = index; i < width * heigth; i += stride)
	{
		destImage[4 * i + 0] = srcImage[i * 3 + 0];
		destImage[4 * i + 1] = srcImage[i * 3 + 1];
		destImage[4 * i + 2] = srcImage[i * 3 + 2];
		destImage[4 * i + 3] = Clamp((int)alpha, 0, 255);
	}
}


void hav_nv12_bgra32_SDR(unsigned char* SDRLuma,
	unsigned char* SDRChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float wr,
	float wb,
	unsigned char* rgbImage,
	bool exAlpha,
	unsigned int alpha
)
{
	float wg = 1 - wr - wb;
	float wgb = -wb * (1.0f - wb) / 0.5f / (1 - wb - wr);
	float wgr = -wr * (1 - wr) / 0.5f / (1 - wb - wr);
	float white_black_coeff = 1.16f;

	float wr_coef = (1.0f - wr) / 0.5f;
	float wb_coef = (1.0f - wb) / 0.5f;

	nv12_SDR_bgra32_SDR_kernel << <320, 180 >> > (SDRLuma,
		SDRChroma,
		width,
		heigth,
		inverted,
		wr,
		wb,
		wg,
		wgb,
		wgr,
		wr_coef,
		wb_coef,
		white_black_coeff,
		rgbImage,
		true,
		alpha);
}

void hav_p016_HDR10_bgra64_HDR10_PQ_ACES(unsigned short* HDRLuma, 
	unsigned short* HDRChroma, 
	unsigned int width, 
	unsigned int heigth, 
	bool inverted,
	float RGB_XYZ_Matrix[3][3],
	float XYZ_RGB_Matrix[3][3],
	float max_content_luminance, 
	float display_luminance, 
	unsigned short* HDRRGBA, 
	bool exAlpha, 
	unsigned int alpha)
{
	float wr = RGB_XYZ_Matrix[1][0];
	float wb = RGB_XYZ_Matrix[1][2];
	float wg = 1.0f - wr - wb;
	float wgb = -wb * (1.0f - wb) / 0.5f / wg;
	float wgr = -wr * (1 - wr) / 0.5f / wg;
	float white_black_coeff = 1.16f;

	float wr_coef = (1.0f - RGB_XYZ_Matrix[1][0]) / 0.5f;
	float wb_coef = (1.0f - RGB_XYZ_Matrix[1][2]) / 0.5f;

	//no check
	hipMemcpyToSymbol(HIP_SYMBOL(__device_RGB_XYZ_Mat), RGB_XYZ_Matrix, 9 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(__device_XYZ_RGB_Mat), XYZ_RGB_Matrix, 9 * sizeof(float));

	p016_HDR10_bgra64_HDR10_PQ_ACES_kernel << <1240, 360 >> > (HDRLuma,
		HDRChroma,
		width,
		heigth,
		inverted,
		max_content_luminance,
		display_luminance,
		XYZ_RGB_Matrix,
		RGB_XYZ_Matrix,
		wr,
		wb,
		wg,
		wgb,
		wgr,
		wr_coef,
		wb_coef,
		1.16f,
		HDRRGBA,
		exAlpha,
		alpha);
}


void hav_p016_HDR10_bgra64_HDR10_PQ_Reinhard(unsigned short* HDRLuma,
	unsigned short* HDRChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float RGB_XYZ_Matrix[3][3],
	float XYZ_RGB_Matrix[3][3],
	float max_content_luminance,
	float display_luminance,
	unsigned short* HDRRGBA,
	bool exAlpha,
	unsigned int alpha)
{
	float wr = RGB_XYZ_Matrix[1][0];
	float wb = RGB_XYZ_Matrix[1][2];
	float wg = 1.0f - wr - wb;
	float wgb = -wb * (1.0f - wb) / 0.5f / wg;
	float wgr = -wr * (1 - wr) / 0.5f / wg;
	float white_black_coeff = 1.16f;

	float wr_coef = (1.0f - RGB_XYZ_Matrix[1][0]) / 0.5f;
	float wb_coef = (1.0f - RGB_XYZ_Matrix[1][2]) / 0.5f;

	//no check
	hipMemcpyToSymbol(HIP_SYMBOL(__device_RGB_XYZ_Mat), RGB_XYZ_Matrix, 9 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(__device_XYZ_RGB_Mat), XYZ_RGB_Matrix, 9 * sizeof(float));

	p016_HDR10_bgra64_HDR10_PQ_Reinhard_kernel << <1240, 360 >> > (HDRLuma,
		HDRChroma,
		width,
		heigth,
		inverted,
		max_content_luminance,
		display_luminance,
		wr,
		wb,
		wg,
		wgb,
		wgr,
		wr_coef,
		wb_coef,
		1.16f,
		HDRRGBA,
		exAlpha,
		alpha);
}

void hav_p016_HDR10_bgra64_HDR10_Linear(unsigned short* HDRLuma,
	unsigned short* HDRChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float display_luminance,
	float wr,
	float wb,
	unsigned short* HDRRGBA,
	bool exAlpha,
	unsigned int alpha)
{
	float wg = 1.0f - wr - wb;
	float wgb = -wb * (1.0f - wb) / 0.5f / (1 - wb - wr);
	float wgr = -wr * (1 - wr) / 0.5f / (1 - wb - wr);
	float white_black_coeff = 1.16f;

	float wr_coef = (1.0f - wr) / 0.5f;
	float wb_coef = (1.0f - wb) / 0.5f;

	p016_HDR10_bgra64_HDR10_Linear_kernel << <1240, 360 >> > (HDRLuma,
		HDRChroma,
		width,
		heigth,
		inverted,
		wr,
		wb,
		wg,
		wgb,
		wgr,
		wr_coef,
		wb_coef,
		display_luminance,
		1.16f,
		HDRRGBA,
		exAlpha,
		alpha);
}

void hav_p016_HDR10_bgra32_SDR_Linear(unsigned short* cuLuma,
	unsigned short* cuChroma,
	unsigned int width,
	unsigned int heigth,
	bool inverted,
	float wr,
	float wb,
	unsigned char* destImage,
	bool exAlpha,
	unsigned int alpha
)
{
	float wg = 1.0f - wr - wb;
	float wgb = -wb * (1.0f - wb) / 0.5f / (1 - wb - wr);
	float wgr = -wr * (1 - wr) / 0.5f / (1 - wb - wr);
	float white_black_coeff = 1.16f;

	float wr_coef = (1.0f - wr) / 0.5f;
	float wb_coef = (1.0f - wb) / 0.5f;

	p016_HDR10_bgra32_SDR_Linear_kernel << <1240, 360 >> > (cuLuma,
		cuChroma,
		width,
		heigth,
		inverted,
		wr,
		wb,
		wg,
		wgb,
		wgr,
		wr_coef,
		wb_coef,
		white_black_coeff,
		destImage,
		exAlpha,
		alpha
	);
}

void hav_bgr24_bgra32_SDR(unsigned char* bgr, unsigned int width, unsigned int height, unsigned int alpha, unsigned char* bgra)
{
	bgr24_bgra32_kernel << <320, 180 >> > (bgr, width, height, alpha, bgra);

}

